
#include <hip/hip_runtime.h>
__global__ void tanh_kernel(float* Y,
                            int batch_size,
                            int num){
    int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    Y[idx] = tanh(Y[idx]);
    __syncthreads();
}

void launch_tanh(float* Y,
                int batch_size,
                int num){
    dim3 gridSize((batch_size * num + 1023)/ 1024);
    dim3 blockSize(1024);
    tanh_kernel<<<gridSize, blockSize>>>(Y, batch_size, num);
}